#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <time.h>

// includes, kernels
__constant__ float n_c; // allocation on the kernel
#include "vector_dot_product_kernel.cu"

void run_test(unsigned int);
float compute_on_device(float *, float *,int);
void check_for_error(char *);
extern "C" float compute_gold( float *, float *, unsigned int);

int 
main( int argc, char** argv) 
{
	if(argc != 2){
		printf("Usage: vector_dot_product <num elements> \n");
		exit(0);	
	}
	unsigned int num_elements = atoi(argv[1]);
	run_test(num_elements);
	return 0;
}

void 
run_test(unsigned int num_elements) 
{
	// Obtain the vector length
	unsigned int size = sizeof(float) * num_elements;

	// Allocate memory on the CPU for the input vectors A and B
	float *A = (float *)malloc(size);
	float *B = (float *)malloc(size);
	
	// Randomly generate input data. Initialize the input data to be floating point values between [-.5 , 5]
	printf("Generating random vectors with values between [-.5, .5]. \n");	
	srand(time(NULL));
	for(unsigned int i = 0; i < num_elements; i++){
		A[i] = (float)rand()/(float)RAND_MAX - 0.5;
		B[i] = (float)rand()/(float)RAND_MAX - 0.5;
	}
	
	printf("Generating dot product on the CPU. \n");
	float reference = compute_gold(A, B, num_elements);
    
	/* Edit this function to compute the result vector on the GPU. 
       The result should be placed in the gpu_result variable. */
	float gpu_result = compute_on_device(A, B, num_elements);

	printf("Result on CPU: %f, result on GPU: %f. \n", reference, gpu_result);
    printf("Epsilon: %f. \n", fabsf(reference - gpu_result));

	// cleanup memory
	free(A);
	free(B);
	
	return;
}

/* Edit this function to compute the dot product on the device using atomic intrinsics. */
float 
compute_on_device(float *A_on_host, float *B_on_host, int num_elements)
{
   float * A_on_device=NULL; 
	float * B_on_device=NULL; 
	float * C_on_device=NULL; 
	float * result = NULL;
 
	//allocate space on the GPU globabl memory 
	hipMalloc((void**)&A_on_device, num_element*sizeof(float)); 
	hipMalloc((void**)&B_on_device, num_element*sizeof(float)); 
	hipMalloc((void**)&C_on_device, sizeof(float)); 

	//copy the values over to GPU
	hipMemcpy(A_on_device, A_on_host, num_element*sizeof(float), hipMemcpyHostToDevice); 
	hipMemcpy(B_on_device, B_on_host, num_element*sizeof(float), hipMemcpyHostToDevice); 

	//set up grid and TB
	int max_TB_size=1024; 
	int max_grid_size=20; 
	int TB_size, num_TB; 
	if(num_elements < max_TB_size){
		TB_size=num_elements; 
		num_TB=1; 
	}else {
		TB_size=max_TB_size; 
		num_TB=num_elements/TB_size; 
		if(num_elements%TB_size>0)
			num_TB++; 
		if(num_TB>max_grid_size)
			num_TB=max_grid_size; 
	}

	dim3 thread_block(TB_size); 
	dim3 grid(num_TB); 
	
	printf("performing vector dot product on the GPU using shared memory and a constant \n");
	struct timeval start, stop; 
	gettimeofday(&start, NULL);

	//copy the constant to GPU
	hipMemcpyToSymbol(HIP_SYMBOL(n_c), num_elements, sizeof(int)); 
	
	//launch the kernel
	vector_dot_product<<grid, thread_block>>(A_on_device, B_on_device, C_on_device); 
	hipDeviceSynchronize();
	check_for_error("KERNEL FAILURE");


	gettimeofday(&stop, NULL);
	printf("Execution time = %fs. \n", (float)(stop.tv_sec - start.tv_sec+\\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

	//copy answer
	hipMemcpy(result, C_on_device, sizeof(float), hipMemcpyDeviceToHost);

	//free up the GPU memory 
	hipFree(A_on_device);
	hipFree(B_on_device); 
	hipFree(C_on_device);  	
	 return result;
}
 
// This function checks for errors returned by the CUDA run time
void 
check_for_error(char *msg)
{
	hipError_t err = hipGetLastError();
	if(hipSuccess != err){
		printf("CUDA ERROR: %s (%s). \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
} 
