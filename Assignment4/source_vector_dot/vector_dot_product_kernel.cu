#include "hip/hip_runtime.h"
#ifndef _VECTOR_DOT_PRODUCT_KERNEL_H_
#define _VECTOR_DOT_PRODUCT_KERNEL_H_

/* Edit this function to complete the functionality of dot product on the GPU using atomics. 
	You may add other kernel functions as you deem necessary. 
 */

__global__ void vector_dot_product(float* Ad, float* Bd, float* Cd)
{
	int k=blockDim.x*gridDim.x; 
	int tid=threadIdx.x+(blockDim.x+blockIdx.x); 
    extern __shared__ double C_shared[];
	int num_strides = *(n_c)/k; 
	if (*(n_c)%k>0)
		num_strides++; 
	C_shared[tid]=0; // C is the size the number of threads 
	int i; 
	for(i=0; i<num_strides; i++)
		if(tid+(k*i)<*(n_c))
			C_shared[tid]+=((double)Ad[tid+(k*i)]*(double)Bd[tid+(k*i)]); 
	 /*Now everything is multiplied and loaded into share d memory that is the
 	* size of k number of threads, and reduction needs to be applied to get the
 	* answer*/
	__syncthreads();
	int depth=1; 
	int stride; 
	for(i=1; i<k; i*=2){
		stride=k/(2*depth); 
		if(tid<stride)
			C_shared[tid]+=C_shared[tid+stride]; 
		depth++; 
		__syncthreads();	
	}
	*Cd=(float)C_shared[0]; //copy back to global memory

}




#endif // #ifndef _VECTOR_DOT_PRODUCT_KERNEL_H
