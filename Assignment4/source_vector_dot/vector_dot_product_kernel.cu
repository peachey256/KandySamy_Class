#include "hip/hip_runtime.h"
#ifndef _VECTOR_DOT_PRODUCT_KERNEL_H_
#define _VECTOR_DOT_PRODUCT_KERNEL_H_

/* Edit this function to complete the functionality of dot product on the GPU using atomics. 
	You may add other kernel functions as you deem necessary. 
 */
__constant__ int n_c[1]; // allocation on the kernel

__global__ void vector_dot_product(float* Ad, float* Bd, float* Cd)
{
	int k=THREAD_COUNT; //blockDim.x*gridDim.x; 
	int tid=threadIdx.x+(blockDim.x*blockIdx.x); 
    __shared__ double C_shared[THREAD_COUNT];
    int n_c_local = n_c[0];
	int num_strides = n_c_local/k; 
	if (n_c[0]%k>0)
		num_strides++; 
    if(tid < n_c_local)
	    C_shared[tid]=0; // C is the size the number of threads 
	int i; 
	for(i=0; i<num_strides; i++)
		if(tid<THREAD_COUNT)
			if((tid+(k*i))<n_c_local)
				C_shared[tid]+=((double)Ad[tid+(k*i)]*(double)Bd[tid+(k*i)]); 

	 /*Now every thing is multiplied and loaded into share d memory that is the
 	* size of k number of threads, and reduction needs to be applied to get the
 	* answer*/  
	__syncthreads(); 
	int stride; 
	for(stride=k; stride>0; stride/=2){
		if(tid<stride && tid+stride < k)
			C_shared[tid]+=C_shared[tid+stride]; 
		__syncthreads();	
	}
	if (tid==0)
		*Cd=(float)C_shared[0]; //copy back to global memory

}




#endif // #ifndef _VECTOR_DOT_PRODUCT_KERNEL_H
