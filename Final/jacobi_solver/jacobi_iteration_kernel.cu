#include "hip/hip_runtime.h"
#include "jacobi_iteration.h"

// Write the GPU kernel to solve the Jacobi iterations
__global__ void jacobi_iteration_kernel (float * Ad, float * Bd, float * xd, float * diff)
{
	    // allocate some shared memory
    __shared__ double A_shared[TILE_SIZE][TILE_SIZE];
    __shared__ double x_shared[TILE_SIZE]; 
	
    // locate yo self within tile - TB and tile size are the same. 
    const unsigned int tileCol = threadIdx.x;
    const unsigned int tileRow = threadIdx.y;

    // locate yo self within Array
    //  ... only true for first file
    const unsigned int row = blockDim.x * blockIdx.x + tileRow;
    const unsigned int col = blockDim.x * blockIdx.x + tileCol;

    // number of tiles we're going to need
    // ... add an extra if not evently divisble
    double partSum = Bd[col];
    int temp;
    int k;

    // moves tile across matrix
    for(k=0; k<MATRIX_SIZE; k+=TILE_SIZE) {
        // check M edge conditions for this tile
        if(k + tileCol < MATRIX_SIZE && row < MATRIX_SIZE)
            A_shared[tileRow][tileCol] = Ad[row*MATRIX_SIZE + k + tileCol];
        else
            A_shared[tileRow][tileCol] = 0.0f;

        if (k + tileCol < MATRIX_SIZE)
            x_shared[tileCol] = Xd[k+tileCol];
        else
            x_shared[tileCol] = 0.0f;

        __syncthreads();

			//NEED to implement with reduction later. 
        for(temp = 0; temp < TILE_SIZE; temp++)
				if(temp!=tileRow)//subtract out all the non diagonal elements 
            	partSum -= A_shared[tileRow][temp] * X_shared[temp];

        __syncthreads();
    }

    if (row < MATRIX_SIZE){
		*diff = (Xd[row]-partSum)*( Xd[row]-partSum); 	
    	Xd[row] = (float)partSum/A_shared[tileRow][tileRow];
		//everything is divided by the diagonal element 
	}



}



