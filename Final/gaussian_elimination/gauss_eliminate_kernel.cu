#include "hip/hip_runtime.h"
 /* Device code. */
#include "gauss_eliminate.h"

__global__ void gauss_division_kernel(float *A, int k)
{
	int tid=k+1+(blockDim.x*blockIdx.x+threadIdx.x); 
	int number_of_updates = MATRIX_SIZE-k-1; //dont do anything before k. 	
	int n_threads=GRID_SIZE*BLOCK_SIZE; 
	int num_stride=number_of_updates/(n_threads); 
	if(number_of_updates%(n_threads))
		num_strides++;
	int stride;  
	for(stride=0; stride<num_strides; strides+=n_threads)
		if((tid+stride)<MATRIX_SIZE)
			A[k*MATRIX_SIZE+(tid+stride)]/=A[k*MATRIX_SIZE+k]; 
	//at this point all the elements in row k after col k are divided by the
	//value at row k, col k. still have to set k=1... 
}



__global__ void gauss_eliminate_kernel(float *A, int k)
{
	
}

